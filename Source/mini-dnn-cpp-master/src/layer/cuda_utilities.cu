#include "hip/hip_runtime.h"
#include <iostream>
#include "cuda_utilities.h"
#include "../../config.h"

#define TILE_WIDTH 32

#define CHECK(call)                                                \
	{                                                              \
		const hipError_t error = call;                            \
		if (error != hipSuccess)                                  \
		{                                                          \
			fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
			fprintf(stderr, "code: %d, reason: %s\n", error,       \
					hipGetErrorString(error));                    \
			exit(EXIT_FAILURE);                                    \
		}                                                          \
	}

struct GpuTimer
{
	hipEvent_t start;
	hipEvent_t stop;

	GpuTimer()
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
	}

	~GpuTimer()
	{
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}

	void Start()
	{
		hipEventRecord(start, 0);
		hipEventSynchronize(start);
	}

	void Stop()
	{
		hipEventRecord(stop, 0);
	}

	float Elapsed()
	{
		float elapsed;
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed, start, stop);
		return elapsed;
	}
};

static GpuTimer timer;
void startTimer()
{
    timer.Start();
}

float stopTimer()
{
    timer.Stop();

	return timer.Elapsed();
}

__host__ __device__ int idx1D(int r, int c, int colSz) // Create two verision: __host__ to be callable from CPU and run on CPU, __device__ to be callable from GPU and run on GPU
{
    return r * colSz + c;
}

__host__ __device__ int idx1D_col(int r, int c, int rowSz) // Create two verision: __host__ to be callable from CPU and run on CPU, __device__ to be callable from GPU and run on GPU
{
    return c * rowSz + r;
}

__global__ void unrollKernel_1(int C, int H, int W, int K, float* image, float* data_col)
{
	int c, s, h_out, w_out, h_unroll, w_unroll, w_base, p, q;
	int t = blockIdx.x * blockDim.x + threadIdx.x;
	int H_out = H - K + 1;
	int W_out = W - K + 1;
	int W_unroll = H_out * W_out;

	if (t < C * W_unroll)
	{
		c = t / W_unroll;
		s = t % W_unroll;
		h_out = s / W_out;
		w_out = s % W_out;
		h_unroll = h_out * W_out + w_out;
		w_base = c * (K * K);

		for (p = 0; p < K; p++)
		{
			for (q = 0; q < K; q++)
			{
				w_unroll = w_base + p * K + q;
				data_col[w_unroll * W_unroll + h_unroll] = image[c * H * W + (h_out + p) * W + (w_out + q)];
			}
		}
	}
}

__global__ void matrixMultiplicationKernel_1(float* A, float* B, float* C, int m, int n, int k, int image)
{
    // Xác định chỉ số hàng và cột trong ma trận kết quả
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < k) {
        float sum = 0.0f;

        // Tính toán tích vô hướng của hàng A và cột B
        for (int i = 0; i < n; i++) {
            sum += A[row * n + i] * B[i * k + col];
        }

        // Ghi kết quả vào ma trận C
        C[row * k + col] = sum;
    }
}

__global__ void matrixMultiplicationKernel_2(float* A, float* B, float* C, int m, int n, int k, int image)
{
	__shared__ float s_A[TILE_WIDTH][TILE_WIDTH];
	__shared__ float s_B[TILE_WIDTH][TILE_WIDTH];

    int numStride = (n - 1) / TILE_WIDTH + 1;
    int tidY = blockIdx.y * blockDim.y + threadIdx.y;
    int tidX = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;

    for (int stride = 0; stride < numStride; stride++)
    {   
        int globalAIdx = idx1D_col(tidY, stride * TILE_WIDTH + threadIdx.x, m);
        int globalBIdx = idx1D_col(stride * TILE_WIDTH + threadIdx.y, tidX, n);

        if (tidY < m && stride * TILE_WIDTH + threadIdx.x < n)
            s_A[threadIdx.y][threadIdx.x] = A[globalAIdx];
        else
            s_A[threadIdx.y][threadIdx.x] = 0;

        if ((stride * TILE_WIDTH + threadIdx.y) < n && tidX < k)
            s_B[threadIdx.y][threadIdx.x] = B[globalBIdx];
        else
            s_B[threadIdx.y][threadIdx.x] = 0;

        __syncthreads();
        
        for (int i = 0; i < TILE_WIDTH; i++)
        {
            sum += s_A[threadIdx.y][i] * s_B[i][threadIdx.x];
			//if (tidY == 4 && tidX == 0 && image == 2) printf("s_A[%d][%d] = %f, s_B[%d][%d] = %f\n", threadIdx.y, i, s_A[threadIdx.y][i], i, threadIdx.x, s_B[i][threadIdx.x]);
        }
        // __syncthreads();
    }

    if ( (tidY < m) && (tidX < k)) 
		C[idx1D_col(tidY, tidX, m)] = sum;
}

void matrixMultiplicationCPU(float* A, float *B, float *C, int m, int n, int k)
{	
	for (int r = 0; r < m; r++)
        {
            for (int c = 0; c < k; c++)
            {
                for (int i = 0; i < n; i++) 
                {
                    C[idx1D(r, c, k)] += A[idx1D(r, i, n)] * B[idx1D(i, c, k)];
                }
            }
        }
}

void unrollGPUWrapper(int C, int H, int W, int K, float* image, float* data_col)
{
	int H_out = H - K + 1;
	int W_out = W - K + 1;
	int W_unroll = H_out * W_out;
	int num_threads = C * H_out * W_out;
	int block_size = 1024;
	int num_blocks = ceil((float)num_threads / block_size);
	
	// Copy image to device
	float* d_image;
	CHECK(hipMalloc(&d_image, C * H * W * sizeof(float)));
	CHECK(hipMemcpy(d_image, image, C * H * W * sizeof(float), hipMemcpyHostToDevice));

	// Copy data_col to device
	float* d_data_col;
	CHECK(hipMalloc(&d_data_col, C * K * K * W_unroll * sizeof(float)));

	unrollKernel_1<<<num_blocks, block_size>>>(C, H, W, K, d_image, d_data_col);
	CHECK(hipGetLastError());

	// Copy data_col back to host
	CHECK(hipMemcpy(data_col, d_data_col, C * K * K * W_unroll * sizeof(float), hipMemcpyDeviceToHost));
	// Free memory
	CHECK(hipFree(d_image));
	CHECK(hipFree(d_data_col));
}

void matrixMultiplicationGPUWrapper(float* A, float *B, float *C, int m, int n, int k, int i, bool isOptimized)
{	
	memset(C, 0, m * k * sizeof(float));

	dim3 blockSize(32, 32);
	float *d_A, *d_B, *d_C;
	const int size_A = m * n * sizeof(float);
	const int size_B = n * k * sizeof(float);
	const int size_C = m * k * sizeof(float);
	CHECK(hipMalloc(&d_A, size_A));
	CHECK(hipMalloc(&d_B, size_B));
	CHECK(hipMalloc(&d_C, size_C));

	CHECK(hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_B, B, size_B, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_C, C, size_C, hipMemcpyHostToDevice));

	dim3 gridSize( (k - 1)/(blockSize.x) + 1, ( m - 1)/(blockSize.y) + 1);
	if (!isOptimized){
		matrixMultiplicationKernel_1<<<gridSize, blockSize>>>(d_A, d_B, d_C, m, n, k, i);
	}
	else{
		matrixMultiplicationKernel_2<<<gridSize, blockSize>>>(d_A, d_B, d_C, m, n, k, i);
	}
	CHECK(hipGetLastError());

	CHECK(hipMemcpy(C, d_C, size_C, hipMemcpyDeviceToHost));

	CHECK(hipFree(d_A));
	CHECK(hipFree(d_B));
	CHECK(hipFree(d_C));
}